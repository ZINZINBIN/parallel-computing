/**
 * @file main.cpp
 * @author zinzinbin
 * @brief Example Code on lecture 09 : cuda and openacc example
 * @version 0.1
 * @date 2022-05-06
 *
 * How to execute
 * (1) nvcc main.cu -o main.out
 * (2) ./main.out
 */


#include <hip/hip_runtime.h>
#include <iostream>
#define nTx 8
#define nTy 8
using namespace std;

// 자체 실습 : Data transfer, Matrix transpose (CUDA, openacc)
// kernel definintion : subroutines executing on GPUs
__global__ void kernel(void){
    
}

__global__ void VecAdd(float *A, float *B, float *C){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    C[idx] = A[idx] + B[idx];
    printf("c[%d] = %.3f + %.3f = %.3f\n",idx,A[idx], B[idx], C[idx]);
}

__global__ void simpleMultiply(float *a, float *b, float *c, int M, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    double sum = 0;
    for(int i = 0; i < M; i++){
        sum += a[row * M + i] * b[i*N + col];
    }
    c[row *N + col] = sum;
}

__global__ void MatrixTranspose(int *inp_mat, int *out_mat, int Nrow, int Ncol){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;
    out_mat[j * Nrow + i] = inp_mat[Ncol * i + j];
}

__global__ void MatrixTransposeTiled(int *inp_mat, int *out_mat, int Nrow, int Ncol){
    __shared__ int tile[nTx][nTy];
    int i = threadIdx.x;
    int j = threadIdx.y;

    int bx = blockIdx.x * blockDim.x;
    int by = blockIdx.y * blockDim.y;

    tile[i][j] = inp_mat[(bx + i) * Ncol + by + j];
    __syncthreads(); // block 내 모든 thread를 동기화
    out_mat[(by + j) * Nrow + bx + i] = tile[i][j];
}

void print_matrix(int *A, int nrow, int ncol)
{
    int idx;
    for(int i = 0; i < nrow; i++){
        for(int j = 0; j < ncol; j++){
            idx = ncol * i + j;
            printf("%3d ",A[idx]);
        }
        cout << endl;
    }
}

void print_array(int *A, int N){
    for(int i = 0; i < N; i++){
        printf("%3d ", A[i]);
    }
    cout << endl;
}

int main(void){
    // Data transfer
    int N = 32;
    size_t size = N * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *d_a;

    // example 1. cuda memory allocation, copy and deallocation
    cout << "cuda memory allocation" << endl;
    hipMalloc(&d_a, size);

    kernel <<<1,1>>>();

    cout << "cuda memory copy" << endl;
    hipMemcpy(d_a, h_a, N, hipMemcpyHostToDevice);

    hipMemcpy(h_a, d_a, N, hipMemcpyDeviceToHost);

    cout << "cuda memory deallocation" << endl;
    hipFree(d_a);
    free(h_a);

    // example 2. vector addition
    size_t vec_size = N * sizeof(float);
    float *a = (float *)malloc(vec_size);
    float *b = (float *)malloc(vec_size);
    float *c = (float *)malloc(vec_size);

    // float *d_a;
    float *d_b;
    float *d_c;

    for(int i = 0; i < N; i++){
        a[i] = i;
        b[i] = 2 * i;
        c[i] = 0;
    }

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, size, hipMemcpyHostToDevice);

    VecAdd <<<2,N>>> (d_a, d_b, d_c);

    hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
    hipMemcpy(b, d_b, size, hipMemcpyDeviceToHost);
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
        cout << "c[" << i << "] = " << c[i] << endl;
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    // example 3. Matrix Transpose
    int Nrow = 16;
    int Ncol = 16;
    size_t mat_size = Nrow * Ncol * sizeof(int);
    int *inp_mat_h = (int *)malloc(mat_size);
    int *out_mat_h = (int *)malloc(mat_size);
    int *inp_mat_d;
    int *out_mat_d;

    int nBx = Nrow / nTx + (Nrow % nTx != 0);
    int nBy = Ncol / nTy + (Ncol % nTy != 0);
    dim3 grid(nBx, nBy); // nBx x nBy 형태의 block들로 구성된 grid
    dim3 threads(nTx,nTy);

    for(int i = 0; i < Nrow * Ncol; i++){
        inp_mat_h[i] = i;
    }

    hipMalloc(&inp_mat_d, mat_size);
    hipMalloc(&out_mat_d, mat_size);

    hipMemcpy(inp_mat_d, inp_mat_h, mat_size, hipMemcpyHostToDevice);

    MatrixTranspose <<<grid, threads>>> (inp_mat_d, out_mat_d, Nrow, Ncol);

    hipMemcpy(out_mat_h, out_mat_d, mat_size, hipMemcpyDeviceToHost);
    
    cout << "# input matrix" << endl;
    print_matrix(inp_mat_h, Nrow, Ncol);

    cout <<"# output matrix" << endl;
    print_matrix(out_mat_h, Nrow, Ncol);

    // example 4. matrix transpose with shared variable tile
    MatrixTransposeTiled <<<grid, threads>>>(inp_mat_d, out_mat_d, Nrow, Ncol);

    hipMemcpy(out_mat_h, out_mat_d, mat_size, hipMemcpyDeviceToHost);

    cout << "# output matrix for example 4" << endl;
    print_matrix(out_mat_h, Nrow, Ncol);

    hipFree(inp_mat_d);
    hipFree(out_mat_d);
    free(inp_mat_h);
    free(out_mat_h);

    // example 5. openacc
    N = 8;
    size_t a_size = N * sizeof(int);
    int *x = (int *)malloc(a_size);
    int *y = (int *)malloc(a_size);

    #pragma acc kernels
    {
        for(int i = 0; i < N; i++){
            x[i] = i;
            y[i] = 2 * i;
        }
        for(int i = 0; i < N; i++){
            y[i] += 3.0 * x[i];
        }
    }

    cout << "example 5 code" << endl;
    print_array(y,N);

    free(x);
    free(y);

    // matrix transpose with openacc
    int a_row = 16;
    int a_col = 16;
    int *A = new int [a_row * a_col];
    int *B = new int [a_row * a_col];

    for(int i = 0; i < a_row; i++){
        for(int j = 0; j < a_col; j++){
            int idx = i * a_col + j;
            A[idx] = i * j - i + j;
        }
    }

    #pragma acc parallel loop
    {
        for(int j = 0; j < a_col; j++){
            #pragma acc loop
            {
                for(int i = 0; i < a_row; i++){
                    B[j * a_col + i] = A[i * a_col + j];
                }
            }
        }
    }

    cout << "matrix transpose using openacc" << endl;
    cout << "original" << endl;
    print_matrix(A, a_row, a_col);

    cout << "transpose" << endl;
    print_matrix(B, a_row, a_col);

    free(A);
    free(B);

    // example 6. data directive of openacc : explicit data transfer
    float err = 1;
    float tol = 0.0001;
    int iter = 0;
    int iter_max = 256;

    a_row = 16;
    a_col = 16;
    float *M = new float[a_row * a_col];
    float *M_new = new float[a_row * a_col];

    for (int i = 0; i < a_row; i++)
    {
        for (int j = 0; j < a_col; j++)
        {
            int idx = i * a_col + j;
            M[idx] = 2.0 * i + 3.0 * j;
        }
    }

    #pragma acc data copy(M), create(M_new)
    {
        while(err > tol && iter < iter_max){
            iter += 1;
            err = 0;
            #pragma acc parallel loop reduction(max:err)
            {
                for(int i = 1; i < a_row - 1; i++){
                    for(int j = 1; j < a_col - 1; j++){
                        M_new[i * a_col + j] = 0.25 * (
                            M[i * a_col + j - 1] +
                            M[i * a_col + j + 1] +
                            M[i * a_col + j - a_col] +
                            M[i * a_col + j + a_col]
                        );
                        err = max(err, abs(M_new[i * a_col + j] - M[i * a_col + j]));
                    }
                }
            }
        }
    }

    cout << "error : " << err << ", iter : " << iter << endl;
    free(M_new);
    free(M);

    return 0;
}