
#include <hip/hip_runtime.h>
#include <iostream>
#define nTx 16
#define nTy 16

// 자체 실습 : Data transfer, Matrix transpose (CUDA, openacc)

// kernel definintion : subroutines executing on GPUs
__global__ void kernel(void){
    
}

__global__ void VecAdd(float *A, float *B, float *C){
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

__global__ void simpleMultiply(float *a, float *b, float *c, int M, int N){
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    double sum = 0;
    for(int i = 0; i < M; i++){
        sum += a[row * M + i] * b[i*N + col];
    }
    c[row *N + col] = sum;
}

int main(void){
    // Data transfer
    int N = 32;
    size_t size = N * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *d_a;
    hipMalloc(&d_a, size);

    // kernel<<<1,1>>>();

    hipMemcpy(d_a, h_a, N, hipMemcpyHostToDevice);

    hipMemcpy(h_a, d_a, N, hipMemcpyDeviceToHost);

    hipFree(d_a);
    free(h_a);

    return 0;
}