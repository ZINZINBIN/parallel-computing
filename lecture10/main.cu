#include "hip/hip_runtime.h"
/**
 * @file main.cpp
 * @author zinzinbin
 * @brief Example Code on lecture 10 : cuda
 * @version 0.1
 * @date 2022-05-13
 *
 * How to execute
 * (1) nvcc main.cu -o main.out
 * (2) ./main.out
 */

#include <iostream>
#define nTx 16
#define nTy 16

__global__ void kernel(void){
    cout << "kernel function proceeded" << endl;
}

int main(int argc, char *argv[])
{
    int N = 32;
    size_t size = N * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *h_v;
    float *d_a;
    float *d_v;

    int nBx = 4;
    int nBy = 4;

    dim3 grid(nBx,nBy);
    dim3 block(nTx,nTy);

    // example 1. asynchronous data transfer
    cout << "cuda memory allocation to host" << endl;
    hipHostAlloc(&d_a, size);

    hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice, 0); // the last argument : stream 
    kernel <<<grid,block>>> ();

    // example 1-1. use cuda stream for asynchronous data transfer
    hipStream_t stream1, stream2;

    hipHostAlloc(&h_v, size, hipHostMallocDefault);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // stream 1 
    hipMemcpyAsync(d_v, h_v, size, hipMemcpyHostToDevice, stream1);

    // stream 2
    kernel <<<grid, block, stream2>>>();

    // free memory
    cudaStreamDistroy(stream1);
    hipStreamDestroy(stream2);
    hipHostFree(h_v);

    // example 1-3. asynchronous data transfer using openacc

    return 0;
}