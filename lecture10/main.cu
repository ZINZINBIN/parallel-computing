#include "hip/hip_runtime.h"
/**
 * @file main.cpp
 * @author zinzinbin
 * @brief Example Code on lecture 10 : cuda
 * @version 0.1
 * @date 2022-05-13
 *
 * How to execute
 * (1) nvcc main.cu -o main.out
 * (2) ./main.out
 */

#include <iostream>
#define nTx 4
#define nTy 4
#define BLOCK_SIZE 16

using namespace std;

__global__ void kernel(void){
    // printf("kernel function proceed\n");
}

__global__ void matmul(float *a, float *b, float *c, int M, int N){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    for (int i = 0; i < M; i++)
    {
        sum += a[row * M + i] * b[i * N + col];
    }
    c[row * N + col] = sum; // C[row, col] = sum of a[row,i] * b[i, col]
}

__global__ void matmul_sm(float *a, float *b, float *c, int L, int M, int N){
    __shared__ float aTile[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float bTile[BLOCK_SIZE][BLOCK_SIZE];

    int brow = blockIdx.x;
    int bcol = blockIdx.y;

    int trow = threadIdx.x;
    int tcol = threadIdx.y;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;

    int nBlock = L / BLOCK_SIZE + (L % BLOCK_SIZE != 0);

    for(int k = 0; k < nBlock; k++){
        
    }

}


int main(int argc, char *argv[])
{
    int N = 8;
    size_t size = N * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *h_v;
    float *d_a;
    float *d_v;

    int nBx = 2;
    int nBy = 2;

    dim3 grid(nBx,nBy);
    dim3 block(nTx,nTy);

    // example 1. asynchronous data transfer
    cout << "cuda memory allocation to host" << endl;
    //hipHostAlloc(&d_a, size);
    hipMalloc(&d_a, size);

    hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice, 0); // the last argument : stream 
    kernel <<<grid,block>>> ();

    // example 1-1. use cuda stream for asynchronous data transfer
    hipStream_t stream1, stream2;

    hipHostAlloc(&h_v, size, hipHostMallocDefault);
    hipMalloc(&d_v, size);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // stream 1 
    hipMemcpyAsync(d_v, h_v, size, hipMemcpyHostToDevice, stream1);

    // stream 2
    kernel <<<grid, block, 0, stream2>>>();

    // free memory
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipHostFree(h_v);

    // example 1-3. asynchronous data transfer using openacc
    int *a = new int[N];
    int *b = new int[N];
    int *c = new int[N];
    
    #pragma acc parallel loop async(1)
    {
        for(int i = 0; i < N; i++){
            a[i] = i % 8;
        }
    }

    #pragma acc parallel loop async(1)
    {
        for (int i = 0; i < N; i++)
        {
            b[i] = 2 * a[i];
        }
    }
    
    #pragma acc wait(1) async(2)
    {
        for (int i = 0; i < N; i++)
        {
            c[i] = a[i] + b[i];
        }
    }

    for(int i = 0; i < N; i++){
        printf("a : %d, b : %d, c : %d", a[i],b[i],c[i]);
        cout<<endl;
    }

    free(a);
    free(b);
    free(c);

    // example 2. Matrix multiplication with simple example and using shared memory
    int M_row = 16;
    int M_col = 16;
    int N_row = 16;
    int N_col = 16;
    float *M = new float[M_row * M_col];
    float *N = new float[N_row * N_col];

    float *M_d;
    float *N_d;

    hipMalloc(&M_d, M_row * M_col);
    hipMalloc(&N_d, N_row * N_col);

    for (int i = 0; i < N_row; i++)
    {
        for (int j = 0; j < N_col; j++)
        {
            int idx = i * N_col + j;
            N[idx] = 2.0 * i + 3.0 * j;
        }
    }

    for (int i = 0; i < M_row; i++)
    {
        for (int j = 0; j < M_col; j++)
        {
            int idx = i * M_col + j;
            M[idx] = 3.0 * i + 2.0 * j;
        }
    }



    return 0;
}