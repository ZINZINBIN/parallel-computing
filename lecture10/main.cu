/**
 * @file main.cpp
 * @author zinzinbin
 * @brief Example Code on lecture 10 : cuda
 * @version 0.1
 * @date 2022-05-13
 *
 * How to execute
 * (1) nvcc main.cu -o main.out
 * (2) ./main.out
 */


#include <hip/hip_runtime.h>
#include <iostream>
#define nTx 4
#define nTy 4
#define BLOCK_SIZE 16

using namespace std;

__global__ void kernel(void){
    // printf("kernel function proceed\n");
}

// matrix multiplication with 1D array
__global__ void matmul(float *a, float *b, float *c, int M, int N){
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;
    for (int i = 0; i < M; i++)
    {
        sum += a[row * M + i] * b[i * N + col];
    }
    c[row * N + col] = sum; // C[row, col] = sum of a[row,i] * b[i, col]
}

// matrix multiplication using shared memory
__global__ void matmul_sm(float *a, float *b, float *c, int L, int M, int N){
    __shared__ float aTile[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float bTile[BLOCK_SIZE][BLOCK_SIZE];

    int brow = blockIdx.x;
    int bcol = blockIdx.y;

    int trow = threadIdx.x;
    int tcol = threadIdx.y;

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    float sum = 0;

    int nBlock = L / BLOCK_SIZE + (L % BLOCK_SIZE != 0);

    for(int k = 0; k < nBlock; k++){
        aTile[trow][tcol] = a[(BLOCK_SIZE*brow + trow)*M + k * BLOCK_SIZE + tcol];
        bTile[trow][tcol] = b[(k * BLOCK_SIZE + trow) * N + BLOCK_SIZE * bcol + tcol];
        __syncthreads();
        for (int i = 0; i < BLOCK_SIZE; i++){
            sum += aTile[trow][i] * bTile[i][tcol];
        }
        __syncthreads();
    }

    if(row < L && col < N){
        c[row * M + col] = sum;
    }
}

__global__ void reduction_sum(float *psum){
    int half = blockDim.x / 2;
    int id = threadIdx.x;

    while(half >= 1){
        if(id < half){
            psum[id] += psum[id + half];
        }
        half /= 2;
    }
}


int main(int argc, char *argv[])
{
    int N = 8;
    size_t size = N * sizeof(float);
    float *h_a = (float *)malloc(size);
    float *h_v;
    float *d_a;
    float *d_v;

    int nBx = 2;
    int nBy = 2;

    dim3 grid(nBx,nBy);
    dim3 block(nTx,nTy);

    // example 1. asynchronous data transfer
    cout << "cuda memory allocation to host" << endl;
    //cudaHostAlloc(&d_a, size);
    hipMalloc(&d_a, size);

    hipMemcpyAsync(d_a, h_a, size, hipMemcpyHostToDevice, 0); // the last argument : stream 
    kernel <<<grid,block>>> ();

    // example 1-1. use cuda stream for asynchronous data transfer
    hipStream_t stream1, stream2;

    hipHostAlloc(&h_v, size, hipHostMallocDefault);
    hipMalloc(&d_v, size);
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // stream 1 
    hipMemcpyAsync(d_v, h_v, size, hipMemcpyHostToDevice, stream1);

    // stream 2
    kernel <<<grid, block, 0, stream2>>>();

    // free memory
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipHostFree(h_v);

    // example 1-3. asynchronous data transfer using openacc
    int *a = new int[N];
    int *b = new int[N];
    int *c = new int[N];
    
    #pragma acc parallel loop async(1)
    {
        for(int i = 0; i < N; i++){
            a[i] = i % 8;
        }
    }

    #pragma acc parallel loop async(1)
    {
        for (int i = 0; i < N; i++)
        {
            b[i] = 2 * a[i];
        }
    }
    
    #pragma acc wait(1) async(2)
    {
        for (int i = 0; i < N; i++)
        {
            c[i] = a[i] + b[i];
        }
    }

    for(int i = 0; i < N; i++){
        printf("a : %d, b : %d, c : %d", a[i],b[i],c[i]);
        cout<<endl;
    }

    free(a);
    free(b);
    free(c);

    // example 2. Matrix multiplication with simple example and using shared memory
    int M_row = 16;
    int M_col = 16;
    int N_row = 16;
    int N_col = 16;
    float *M = new float[M_row * M_col];
    float *N_ = new float[N_row * N_col];

    float *M_d;
    float *N_d;

    hipMalloc(&M_d, M_row * M_col);
    hipMalloc(&N_d, N_row * N_col);

    for (int i = 0; i < N_row; i++)
    {
        for (int j = 0; j < N_col; j++)
        {
            int idx = i * N_col + j;
            N_[idx] = 2.0 * i + 3.0 * j;
        }
    }

    for (int i = 0; i < M_row; i++)
    {
        for (int j = 0; j < M_col; j++)
        {
            int idx = i * M_col + j;
            M[idx] = 3.0 * i + 2.0 * j;
        }
    }


    // example 3. 2D array in Cuda C : memory allocation from host device with 2D array to GPU with 1D array
    int N_size = 32;
    float **example3_A = new float *[N_size];
    float *example3_x = new float [N_size];
    float *example3_y = new float [N_size];

    float *example3_d_A = new float [N_size];
    float *example3_d_x = new float [N_size];
    float *example3_d_y = new float [N_size];

    for(int i = 0; i < N_size; i++){
        example3_A[i] = new float [N_size];
    }

    hipMalloc(&example3_d_A, N_size * N_size * sizeof(float));
    hipMalloc(&example3_d_x, N_size * sizeof(float));
    hipMalloc(&example3_d_y, N_size * sizeof(float));

    hipMemcpy(example3_d_A, example3_A, N_size * N_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(example3_d_x, example3_x, N_size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(example3_d_y, example3_y, N_size * sizeof(float), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    matmul<<<numBlocks,threadsPerBlock>>>(example3_d_A, example3_d_x, example3_d_y, N_size, N_size);
    
    hipMemcpy(example3_y, example3_d_y, N_size * N_size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(example3_d_y);
    hipFree(example3_d_x);
    hipFree(example3_d_A);

    free(example3_A);
    free(example3_x);
    free(example3_y);

    // example 4. Memory allocation and transfer for 2D or 3D array to 1D array in CUDA
    int example4_N = 32;
    float **example4_A = new float *[example4_N];
    float *example4_d_A = new float[example4_N];
 
    for (int i = 0; i < example4_N; i++)
    {
        example4_A[i] = new float[example4_N];
    }

    // cudaMallocPitch(&example4_d_A, &example4_A, N_size * sizeof(float), N_size); // args : &devptr,&devpitch, N_col * sizeof(type), N_row 
    
    // example 5. multiple GPU
    int gpu_num = 0;
    hipGetDeviceCount(&gpu_num);

    cout << "hipGetDeviceCount : " << gpu_num << endl;

    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, 1);

    hipSetDevice(0); // using gpu : 0 only

    // example 6. Zero Copy
    // zero-copy memory는 device memory에 mapping되는 pinned-memory
    // host와 device 모두 zero copy에 접근할 수 있다. 
    // more detail info : https://junstar92.tistory.com/285
    float *a_h; 
    float *a_map;

    int n_size = 32;
    int nBytes = sizeof(float) * n_size;

    hipGetDeviceProperties(&props, 0);

    if(!props.canMapHostMemory){
        exit(0);
    }

    hipSetDeviceFlags(hipDeviceMapHost);
    hipHostAlloc(&a_h, nBytes, hipHostMallocMapped); // cudaHostAllocMapped : mapped pinned-memory, cudaHostAllocPortable : zero-copy between GPUs
    hipHostGetDevicePointer(&a_map, a_h, 0);
    
    // cudaFreeHost();

    // example 7. atomic functions
    // atomic function : shared memory를 이용한 계산을 통해 race conditions을 방지한다
    cout << "atomic functions example" << endl;
    int arr_a[5]= {1,2,3,4,5};
    int *arr_dev_a;
    
    hipMalloc(&arr_dev_a, 5 * sizeof(int));
    hipMemcpy(arr_dev_a, arr_a, 5 * sizeof(int), hipMemcpyHostToDevice);

    int const_c = 5;
    int y;

    // y = atomicAdd(&arr_a[0], const_c);
    // cout << "y = " << y << endl;

    hipFree(arr_dev_a);

    // example 8. Reduction example with OpenACC
    int example8_M = 8;
    int example8_N = 8;
    int example8_L = 8;

    float **example8_A = new float *[example8_M];
    float **example8_B = new float *[example8_N];
    float **example8_C = new float *[example8_M];
    for(int i = 0; i < example8_M; i++){
        example8_A[i] = new float [example8_N];
        example8_C[i] = new float [example8_L];
    }

    for (int i = 0; i < example8_N; i++)
    {
        example8_B[i] = new float[example8_L];
    }

    #pragma acc parallel loop collapse(2)
    {
        for(int i = 0; i < example8_M; i++){
            for(int j = 0; j < example8_N; j++){
                float c_ij = 0;
                #pragma acc loop reduction(+:c_ij)
                {
                    for(int k = 0; k < example8_L; k++){
                        c_ij += example8_A[i][k] * example8_B[k][j];
                    }
                }
                example8_C[i][j] = c_ij;

            }
        }
    }

    // example 9. reduction in shared memory example
    // detail : https://junstar92.tistory.com/290
    
    return 0;
}